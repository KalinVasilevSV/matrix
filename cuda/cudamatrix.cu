#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <iostream>
#include <omp.h>
//#include <CL/cl2.hpp>
//#include <opencl.hpp>

//#define CL_HPP_TARGET_OPENCL_VERSION 300

__global__
void mult(double *A, double *B, double *C, const long size){
   for (long i=0; i < size; i++)
     for (long j=0; j < size; j++){
       C[i*size+j] = 0.0;
       for (long k=0; k < size; k++)
         C[i*size+j] += A[i*size+k] * B[k*size+j];
    }
}

void PrintMatrix(double *A, const long size)
{
  for (long i = 0; i < size; i++) {
    for (long j = 0; j < size; j++)
      printf("%f ", A[i*size+j]);
    printf("\n");
  }
}

void Test()
{
  long size = 3;

  double A[size][size];
  double B[size][size];
  double C[size][size];

  A[0][0] = 14.0; A[0][1] = 9.0;  A[0][2] = 3.0;
  A[1][0] = 2.0;  A[1][1] = 11.0; A[1][2] = 15.0;
  A[2][0] = 0.0;  A[2][1] = 12.0; A[2][2] = 17.0;

  B[0][0] = 12.0; B[0][1] = 25.0; B[0][2] = 5.0;
  B[1][0] = 9.0;  B[1][1] = 10.0; B[1][2] = 0.0;
  B[2][0] = 8.0;  B[2][1] = 5.0;  B[2][2] = 1.0;

//  Multiply((double*)&A, (double*)&B, (double*)&C, size);

  PrintMatrix((double*)&C, size);
}

int main(int argc, char *argv[])
{
//  double start, end;

  if (argc == 1) {
    Test();
  } else {
    long size = atol(argv[1]);
    printf("size=%ld\n", size);
    double *A = new double[size*size];
    double *B = new double[size*size];
    double *C = new double[size*size];

    hipMallocManaged(&A,sizeof(double)*size*size);
    hipMallocManaged(&B,sizeof(double)*size*size);
    hipMallocManaged(&C,sizeof(double)*size*size);

//    start = omp_get_wtime();
    mult<<<1,1>>>(A,B,C,size);
//    end = omp_get_wtime();

    hipDeviceSynchronize();

//    Multiply(A, B, C, size);
    PrintMatrix(C, size);

    hipProfilerStop();

    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
  }
}
